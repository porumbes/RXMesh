#include "hip/hip_runtime.h"
#include "gtest/gtest.h"
#include "rxmesh/space_filling.h"

TEST(RXMesh, SpaceFilling)
{
    using namespace rxmesh;

    unsigned x = 1, y = 0;
    int      order = 1;
    std::cout << x << ", " << y << '\n';
    unsigned d = Hilbert::distance_from(x, y, order);
    std::cout << "d = " << d << '\n';
    EXPECT_TRUE(d == 3);

    unsigned xp=0, yp=0;
    Hilbert::point_from_distance(3, order, &xp, &yp);
    EXPECT_TRUE((xp == x) && (yp == y));


    std::cout << "======Hilbert ==========================\n";
    order = 3;
    unsigned maxd = UINT32_MAX;

    for (unsigned i = 0; i < 1 << order; i++) 
    {
        for (unsigned j = 0; j < 1 << order; j++) 
        {
           std::cout << "-------------\n";
           std::cout << "i = " << i << ", j = " << j << '\n';
            d = Hilbert::distance_from(i, j, order);
           std::cout << "d = " << d << '\n';

            Hilbert::point_from_distance(d, order, &xp, &yp);
            std::cout << "xp = " << xp << ", yp = " << yp << '\n';
            EXPECT_TRUE((xp == i) && (yp == j));
            maxd = std::max(maxd, d);
        }
    }
    std::cout << "maxd = " << maxd << std::endl;
    std::cout << "======End Hilbert ==========================\n";
}
