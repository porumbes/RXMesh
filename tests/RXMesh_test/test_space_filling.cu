#include "hip/hip_runtime.h"
#include "gtest/gtest.h"

namespace Hilbert 
{

// Based on Hacker's Delight 2nd edition
unsigned int distance_from(unsigned int x, unsigned int y, int curve_order)
{
    int      n = curve_order;
    int      i, xi, yi;
    unsigned s, temp;

    s = 0;  // Initialize.

    for (i = n - 1; i >= 0; i--) 
    {
        xi = (x >> i) & 1;  // Get bit i of x.
        yi = (y >> i) & 1;  // Get bit i of y.
        if (yi == 0) 
        {
            temp = x;             // Swap x and y and,
            x    = y ^ (-xi);     // if xi = 1,
            y    = temp ^ (-xi);  // complement them.
        }
        s = 4 * s + 2 * xi + (xi ^ yi);  // Append two bits to s.
    }

    return s;
}

// Based on Hacker's Delight 2nd edition
void point_from_distance(unsigned s, int order, unsigned* xp, unsigned* yp)
{
    int      n = order;
    int      i, sa, sb;
    unsigned x(0), y(0), temp(0);
    for (i = 0; i < 2 * n; i += 2) {
        sa = (s >> (i + 1)) & 1;  // Get bit i+1 of s.
        sb = (s >> i) & 1;        // Get bit i of s.
        if ((sa ^ sb) == 0) {     // If sa,sb = 00 or 11,
            temp = x;             // swap x and y,
            x    = y ^ (-sa);     // and if sa = 1,
            y    = temp ^ (-sa);  // complement them.
        }
        x = (x >> 1) | (sa << 31);         // Prepend sa to x and
        y = (y >> 1) | ((sa ^ sb) << 31);  // (sa^sb) to y.
    }
    *xp = x >> (32 - n);  // Right-adjust x and y
    *yp = y >> (32 - n);  // and return them to
}                         // the caller.

}

TEST(RXMesh, SpaceFilling)
{
    unsigned x = 1, y = 0;
    int      order = 1;
    std::cout << x << ", " << y << '\n';
    unsigned d = Hilbert::distance_from(x, y, order);
    std::cout << "d = " << d << '\n';
    EXPECT_TRUE(d == 3);

    unsigned xp=0, yp=0;
    Hilbert::point_from_distance(3, order, &xp, &yp);
    EXPECT_TRUE((xp == x) && (yp == y));


    std::cout << "======Hilbert ==========================\n";
    order = 3;
    unsigned maxd = UINT32_MAX;

    for (unsigned i = 0; i < 1 << order; i++) 
    {
        for (unsigned j = 0; j < 1 << order; j++) 
        {
           std::cout << "-------------\n";
           std::cout << "i = " << i << ", j = " << j << '\n';
            d = Hilbert::distance_from(i, j, order);
           std::cout << "d = " << d << '\n';

            Hilbert::point_from_distance(d, order, &xp, &yp);
            std::cout << "xp = " << xp << ", yp = " << yp << '\n';
            EXPECT_TRUE((xp == i) && (yp == j));
            maxd = std::max(maxd, d);
        }
    }
    std::cout << "maxd = " << maxd << std::endl;
    std::cout << "======End Hilbert ==========================\n";
}
